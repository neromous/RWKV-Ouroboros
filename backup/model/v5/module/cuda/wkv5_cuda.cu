#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include "ATen/ATen.h"
typedef at::BFloat16 bf16;
typedef at::Half fp16;
typedef float fp32;
#define MM8_ONE_JSPLIT 16
#define MM8_ONE_TILE 256
#define EMBSPLIT 256
#define EMBBLOCK 16

//---------------
//
// Optimized mm8 operations
//
//---------------

template <typename DTYPE>
__global__ void kernelc_mm8_one(
    const unsigned long long N, const unsigned long long M,
    const DTYPE *__restrict__ const x,
    const uint8_t *__restrict__ const w, const unsigned long long w_stride,
    float *__restrict__ const y,
    const float *__restrict__ const r,
    const float *__restrict__ const o,
    const unsigned long long offset,
    unsigned long long tokenlength)
{

    for (unsigned long long token = 0; token < tokenlength; token++)
    {
        const unsigned long long k = blockIdx.y * blockDim.y + threadIdx.y;
        const unsigned long long j0 = min(N, blockIdx.x * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));
        const unsigned long long j1 = min(N, (blockIdx.x + 1) * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));

        if (k < M)
        {
            float y_local = 0;
            for (unsigned long long j = j0; j < j1; ++j)
            {
                y_local += float(x[j + N * token]) * ((w[j * w_stride + k + offset * N * M] * r[j + offset * N] + o[j + offset * N]));
            }
            atomicAdd(reinterpret_cast<float *>(&y[k + M * token]), *reinterpret_cast<float *>(&y_local));
        }
    }
}

void cudac_mm8_one(unsigned long long N, unsigned long long M,
                   float *x,
                   uint8_t *w, unsigned long long w_stride,
                   float *y,
                   float *r,
                   float *o,
                   unsigned long long offset,
                     unsigned long long tokenlength)
{
    dim3 blockSize(1, MM8_ONE_TILE);
    dim3 gridSize(MM8_ONE_JSPLIT, (M + blockSize.y - 1) / blockSize.y);
    kernelc_mm8_one<<<gridSize, blockSize>>>(
        N, M, x, w, w_stride, y, r, o, offset, tokenlength);
}

/*
template <typename F>
__global__ void kernel_forward(const int B, const int T, const int C, const int H,
                               const F *__restrict__ const _r, const F *__restrict__ const _k, const F *__restrict__ const _v, const float *__restrict__ _w, const F *__restrict__ _u,
                               F *__restrict__ const _y)
{
    const int b = blockIdx.x / H;
    const int h = blockIdx.x % H;
    const int i = threadIdx.x;
    _w += h*_N_;
    _u += h*_N_;

    __shared__ float r[_N_], k[_N_], u[_N_], w[_N_];
    float state[_N_] = {0};

    __syncthreads();
    u[i] = float(_u[i]);
    w[i] = float(_w[i]);
    __syncthreads();

    for (int t = b*T*C + h*_N_ + i; t < (b+1)*T*C + h*_N_ + i; t += C)
    {
        __syncthreads();
        r[i] = float(_r[t]);
        k[i] = float(_k[t]);
        __syncthreads();

        const float v = float(_v[t]);
        float y = 0;

        #pragma unroll
        for (int j = 0; j < _N_; j+=4)
        {
            const float4& r_ = (float4&)(r[j]);
            const float4& k_ = (float4&)(k[j]);
            const float4& w_ = (float4&)(w[j]);
            const float4& u_ = (float4&)(u[j]);
            float4& s = (float4&)(state[j]);
            float4 x;

            x.x = k_.x * v;
            x.y = k_.y * v;
            x.z = k_.z * v;
            x.w = k_.w * v;

            y += r_.x * (u_.x * x.x + s.x);
            y += r_.y * (u_.y * x.y + s.y);
            y += r_.z * (u_.z * x.z + s.z);
            y += r_.w * (u_.w * x.w + s.w);

            s.x = s.x * w_.x + x.x;
            s.y = s.y * w_.y + x.y;
            s.z = s.z * w_.z + x.z;
            s.w = s.w * w_.w + x.w;
        }
        _y[t] = F(y);
    }
}
*/

//---------------
//
// Main forward kernels
//
//---------------

template <typename F>
__global__ void kernel_forward_inference(
    const int B, const int T, const int C, const int H,
    float *__restrict__ _state,
    const F *__restrict__ const _r, const F *__restrict__ const _k,
    const F *__restrict__ const _v, const float *__restrict__ _w,
    const F *__restrict__ _u, F *__restrict__ _y
) {
    const int b = blockIdx.x / H;
    const int h = blockIdx.x % H;
    const int i = threadIdx.x;
    _w += h*_N_;
    _u += h*_N_;
    _state += h*_N_*_N_ + i*_N_; // wrong if B > 1 !!!

    __shared__ float r[_N_], k[_N_], u[_N_], w[_N_];

    float state[_N_];
    #pragma unroll
    for (int j = 0; j < _N_; j++)
        state[j] = _state[j];

    __syncthreads();
    u[i] = float(_u[i]);
    w[i] = _w[i];
    __syncthreads();

    for (int t = b*T*C + h*_N_ + i; t < (b+1)*T*C + h*_N_ + i; t += C)
    {
        __syncthreads();
        r[i] = float(_r[t]);
        k[i] = float(_k[t]);
        __syncthreads();

        const float v = float(_v[t]);
        float y = 0;

        #pragma unroll
        for (int j = 0; j < _N_; j+=4)
        {
            const float4& r_ = (float4&)(r[j]);
            const float4& k_ = (float4&)(k[j]);
            const float4& w_ = (float4&)(w[j]);
            const float4& u_ = (float4&)(u[j]);
            float4& s = (float4&)(state[j]);
            float4 x;

            x.x = k_.x * v;
            x.y = k_.y * v;
            x.z = k_.z * v;
            x.w = k_.w * v;

            y += r_.x * (u_.x * x.x + s.x);
            y += r_.y * (u_.y * x.y + s.y);
            y += r_.z * (u_.z * x.z + s.z);
            y += r_.w * (u_.w * x.w + s.w);

            s.x = s.x * w_.x + x.x;
            s.y = s.y * w_.y + x.y;
            s.z = s.z * w_.z + x.z;
            s.w = s.w * w_.w + x.w;
        }
        _y[t] = F(y);
    }

    __syncthreads();
    #pragma unroll
    for (int j = 0; j < _N_; j++)
        _state[j] = state[j];
    __syncthreads();
}

//---------------
//
// Main backwards kernels
//
//---------------

template <typename F>
__global__ void kernel_backward(const int B, const int T, const int C, const int H, float *__restrict__ _state,
    const F *__restrict__ const _r, const F *__restrict__ const _k, const F *__restrict__ const _v, const float *__restrict__ _w, const float *__restrict__ __w, const F *__restrict__ _u, const F *__restrict__ const _gy,
    F *__restrict__ const _gr, F *__restrict__ const _gk, F *__restrict__ const _gv, F *__restrict__ const _gw, F *__restrict__ const _gu)
{
    const int b = blockIdx.x / H;
    const int h = blockIdx.x % H;
    const int i = threadIdx.x;
    _w += h*_N_;
    _u += h*_N_;
    __w += h*_N_;

    __shared__ float w_[_N_], u_[_N_];
    __shared__ float r[_N_], k[_N_], v[_N_], gy[_N_];
    __syncthreads();
    w_[i] = _w[i];
    u_[i] = float(_u[i]);
    __syncthreads();

    const float w = w_[i];
    const float ww = __w[i];
    const float u = u_[i];

    float state[_N_] = {0}, saaaa[_N_] = {0}, sbbbb[_N_] = {0}, scccc[_N_] = {0}, sdddd[_N_] = {0};

    #pragma unroll
    for (int j = 0; j < _N_; j++) {
        state[j] = _state[j];
        // saaaa[j] = _state[j];
        // sbbbb[j] = _state[j];
        // scccc[j] = _state[j];
        // sdddd[j] = _state[j];
    }

    float gw = 0, gu = 0;
    const int t000 = b*T*C + h*_N_ + i;
    const int t111 = (b+1)*T*C + h*_N_ + i;
    const int t222 = t111 - 2*C;

    for (int t = t000; t < t111; t += C)
    {
        __syncthreads();
        v[i] = float(_v[t]);
        gy[i] = float(_gy[t]);
        __syncthreads();

        const float k = float(_k[t]);
        float gr = 0, gu_ = 0;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& s = state[j];
            float x = k * v[j];

            gr += (u * x + s) * gy[j];
            gu_ += x * gy[j];
            s = s * w + x;
        }
        _gr[t] = F(gr);
        gu += float(_r[t]) * gu_;
    }
    _gu[b*C + h*_N_ + i] = F(gu);

    for (int t = t000; t < t222; t += C)
    {
        __syncthreads();
        v[i] = float(_v[t]);
        gy[i] = float(_gy[t + 2*C]);
        __syncthreads();

        const float k = float(_k[t]);
        float gw_ = 0;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& s = saaaa[j];
            float& s2 = sbbbb[j];
            float x = k * v[j];

            float tmp = w * (x + s);
            s = tmp;
            s2 = tmp + w * s2;
            gw_ += s2 * gy[j];
        }
        gw += float(_r[t + 2*C]) * gw_;
    }
    _gw[b*C + h*_N_ + i] = F(ww * gw);

    for (int t = t111 - C; t >= t000; t -= C)
    {
        __syncthreads();
        v[i] = float(_v[t]);
        gy[i] = float(_gy[t]);
        __syncthreads();

        const float rr = float(_r[t]);
        float gk = 0;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& s = scccc[j];
            float x = rr * gy[j];

            gk += (u * x + s) * v[j];
            s = x + s * w;
        }
        _gk[t] = F(gk);
    }

    for (int t = t111 - C; t >= t000; t -= C)
    {
        __syncthreads();
        r[i] = float(_r[t]);
        k[i] = float(_k[t]);
        __syncthreads();

        const float gyy = float(_gy[t]);
        float gv = 0;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& s = sdddd[j];
            float x = gyy * r[j];

            gv += (u_[j] * x + s) * k[j];
            s = x + s * w_[j];
        }
        _gv[t] = F(gv);
    }
}

//---------------
//
// Forward / backward type aliases
//
//---------------

void cuda_forward_bf16(int B, int T, int C, int H, float *state, bf16 *r, bf16 *k, bf16 *v, float *w, bf16 *u, bf16 *y)
{
    assert(H*_N_ == C);
    kernel_forward_inference<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, state, r, k, v, w, u, y);
}
void cuda_forward_fp16(int B, int T, int C, int H, float *state, fp16 *r, fp16 *k, fp16 *v, float *w, fp16 *u, fp16 *y)
{
    assert(H*_N_ == C);
    kernel_forward_inference<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, state, r, k, v, w, u, y);
}
void cuda_forward_fp32(int B, int T, int C, int H, float *state, fp32 *r, fp32 *k, fp32 *v, float *w, fp32 *u, fp32 *y)
{
    assert(H*_N_ == C);
    kernel_forward_inference<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, state, r, k, v, w, u, y);
}

void cuda_backward_bf16(int B, int T, int C, int H, float *state, bf16 *r, bf16 *k, bf16 *v, float *w, float *ww, bf16 *u, bf16 *gy, bf16 *gr, bf16 *gk, bf16 *gv, bf16 *gw, bf16 *gu)
{
    assert(H*_N_ == C);
    assert(_N_%4 == 0);
    kernel_backward<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, state, r, k, v, w, ww, u, gy, gr, gk, gv, gw, gu);
}

void cuda_backward_fp16(int B, int T, int C, int H, float *state, fp16 *r, fp16 *k, fp16 *v, float *w, float *ww, fp16 *u, fp16 *gy, fp16 *gr, fp16 *gk, fp16 *gv, fp16 *gw, fp16 *gu)
{
    assert(H*_N_ == C);
    assert(_N_%4 == 0);
    kernel_backward<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, state, r, k, v, w, ww, u, gy, gr, gk, gv, gw, gu);
}

void cuda_backward_fp32(int B, int T, int C, int H, float *state, fp32 *r, fp32 *k, fp32 *v, float *w, float *ww, fp32 *u, fp32 *gy, fp32 *gr, fp32 *gk, fp32 *gv, fp32 *gw, fp32 *gu)
{
    assert(H*_N_ == C);
    assert(_N_%4 == 0);
    kernel_backward<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, state, r, k, v, w, ww, u, gy, gr, gk, gv, gw, gu);
}